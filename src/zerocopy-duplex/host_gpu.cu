#include "hip/hip_runtime.h"
/* Use half the GPU to write to the host and the other half to read from the
 * host
 */

#include "scope/scope.hpp"

#include "args.hpp"
#include "kernels.hu"

constexpr int rdDimBlock = 256;
constexpr int wrDimBlock = 256;
auto GpuWrFunc = gpu_write<rdDimBlock, int32_t>;
auto GpuRdFunc = gpu_read<rdDimBlock, int32_t>;

#define NAME Comm_ZeroCopy_Duplex_GPUGPU
#define NAME2 "Comm_ZeroCopy_Duplex_GPUGPU"

namespace NAME {
__global__ void busy_wait(clock_t *d, clock_t clock_count) {
  clock_t start_clock = clock64();
  clock_t clock_offset = 0;
  while (clock_offset < clock_count) {
    clock_offset = clock64() - start_clock;
  }
  if (d) {
    *d = clock_offset;
  }
}
} // namespace NAME

auto Comm_ZeroCopy_HostGPU = [](benchmark::State &state, const int numa,
                                const int cuda) {
  numa::ScopedBind binder(numa);

  const size_t pageSize = page_size();

  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  hipStream_t stream[2];
  void *cpu[2] = {};
  hipEvent_t start[2] = {};
  hipEvent_t stop[2] = {};

  OR_SKIP_AND_RETURN(cuda_reset_device(cuda), "");

  cpu[0] = aligned_alloc(pageSize, bytes);
  cpu[1] = aligned_alloc(pageSize, bytes);
  defer(free(cpu[0]));
  defer(free(cpu[1]));
  if (bytes && (!cpu[0] || !cpu[1])) {
    state.SkipWithError(NAME2 " failed to allocate host memory");
    return;
  }
  std::memset(cpu[0], 0xDEADBEEF, bytes);
  std::memset(cpu[1], 0xDEADBEEF, bytes);

  OR_SKIP_AND_RETURN(hipHostRegister(cpu[0], bytes, hipHostRegisterMapped),
                     "");
  OR_SKIP_AND_RETURN(hipHostRegister(cpu[1], bytes, hipHostRegisterMapped),
                     "");
  defer(hipHostUnregister(cpu[0]));
  defer(hipHostUnregister(cpu[1]));

  // get a valid device pointers
  void *dptr[2] = {};
  hipDeviceProp_t prop;
  OR_SKIP_AND_RETURN(hipGetDeviceProperties(&prop, cuda), "");
#if __CUDACC_VER_MAJOR__ >= 9
  if (prop.canUseHostPointerForRegisteredMem) {
#else
  if (false) {
#endif
    dptr[0] = cpu[0];
    dptr[1] = cpu[1];
  } else {
    OR_SKIP_AND_RETURN(hipHostGetDevicePointer(&dptr[0], cpu[0], 0), "");
    OR_SKIP_AND_RETURN(hipHostGetDevicePointer(&dptr[1], cpu[1], 0), "");
  }

  // create streams
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream[0]), "");
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream[1]), "");
  defer(hipStreamDestroy(stream[0]));
  defer(hipStreamDestroy(stream[1]));

  OR_SKIP_AND_RETURN(hipEventCreate(&start[0]), "");
  OR_SKIP_AND_RETURN(hipEventCreate(&start[1]), "");
  defer(hipEventDestroy(start[0]));
  defer(hipEventDestroy(start[1]));
  OR_SKIP_AND_RETURN(hipEventCreate(&stop[0]), "");
  OR_SKIP_AND_RETURN(hipEventCreate(&stop[1]), "");
  defer(hipEventDestroy(stop[0]));
  defer(hipEventDestroy(stop[1]));

  // compute kernel params that are half of GPU
  int rdDimGrid;
  int wrDimGrid;
  {
  int maxActiveBlocks;
  hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
    GpuWrFunc, wrDimBlock, 0);
  wrDimGrid = maxActiveBlocks * prop.multiProcessorCount;
  }
  {
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
      GpuRdFunc, rdDimBlock, 0);
    rdDimGrid = maxActiveBlocks * prop.multiProcessorCount;
    }

  clock_t cycles = 4096;
  for (auto _ : state) {
  restart_iteration:

    // launch the busy-wait kernel
    NAME::busy_wait<<<1, 1, 0, stream[0]>>>(nullptr, cycles);

    // set up the copies
    OR_SKIP_AND_BREAK(hipEventRecord(start[0], stream[0]),
                      ""); // stream 0 start
    gpu_read<rdDimBlock><<<rdDimGrid, rdDimBlock, 0, stream[0]>>>(
        (int32_t *)dptr[0], (int32_t *)nullptr, bytes); // stream 0 copy
    OR_SKIP_AND_BREAK(hipStreamWaitEvent(stream[1], start[0], 0),
                      ""); // stream 1 wait for stream 0 to start
    OR_SKIP_AND_BREAK(hipEventRecord(start[1], stream[1]),
                      ""); // stream 1 start
    gpu_write<wrDimBlock><<<wrDimGrid, wrDimBlock, 0, stream[1]>>>(
        (int32_t *)dptr[1], bytes);                             // stream 1 copy
    OR_SKIP_AND_BREAK(hipEventRecord(stop[1], stream[1]), ""); // stream 1 stop
    OR_SKIP_AND_BREAK(hipStreamWaitEvent(stream[0], stop[1], 0),
                      ""); // stream 0 wait for stream 1 to stop
    OR_SKIP_AND_BREAK(hipEventRecord(stop[0], stream[0]), ""); // stream 0 stop

    // wait for streams to finish work, and restart iteration if needed
    hipError_t err = hipEventQuery(start[0]);
    if (hipSuccess == err) {
      // busy-wait is done, so it was too slow.
      OR_SKIP_AND_BREAK(hipStreamSynchronize(stream[0]), "");
      OR_SKIP_AND_BREAK(hipStreamSynchronize(stream[1]), "");
      cycles *= 2;
      goto restart_iteration;
    } else if (hipErrorNotReady == err) {
      // kernel was long enough
    } else {
      OR_SKIP_AND_BREAK(err, "errored while waiting for kernel");
    }

    OR_SKIP_AND_BREAK(hipEventSynchronize(stop[0]), "");
    float millis = 0;
    OR_SKIP_AND_BREAK(hipEventElapsedTime(&millis, start[0], stop[0]), "");
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes) * 2);
  state.counters["bytes"] = bytes * 2;
  state.counters["numa"] = numa;
  state.counters["cuda"] = cuda;
};

static void registerer() {

  for (auto cuda : unique_cuda_device_ids()) {
    for (auto numa : numa::ids()) {

      std::string name(NAME2);
      name += "/" + std::to_string(numa) + "/" + std::to_string(cuda);
      benchmark::RegisterBenchmark(name.c_str(), Comm_ZeroCopy_HostGPU, numa,
                                   cuda)
          ->ARGS()
          ->UseManualTime();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME2);
