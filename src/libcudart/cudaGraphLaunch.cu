#include "hip/hip_runtime.h"


#include "scope/scope.hpp"

#include "args.hpp"

#define NAME "Comm_cudart_cudaGraphLaunch"

__global__ void Comm_cudart_cudaGraphLaunch_kernel() {}

static void copy_launcher(void *dst, const void *src, const int size,
                          hipStream_t stream, const int iters) {
  for (int i = 0; i < iters; ++i) {
    hipMemcpyAsync(dst, src, size, hipMemcpyDefault, stream);
  }
}

static void kernel_launcher(hipStream_t stream, const int iters) {
  for (int i = 0; i < iters; ++i) {
    Comm_cudart_cudaGraphLaunch_kernel<<<1, 1, 0, stream>>>();
  }
}

auto Comm_cudart_cudaGraphLaunch = [](benchmark::State &state,
                                           const int numa_id,
                                           const int cuda_id) {
  const int launches = state.range(0);

  numa::ScopedBind binder(numa_id);

  OR_SKIP_AND_RETURN(cuda_reset_device(cuda_id), "failed to reset device");
  OR_SKIP_AND_RETURN(hipSetDevice(cuda_id), "failed to set CUDA dst device");

  hipGraph_t graph;
  hipGraphExec_t instance;
  hipStream_t stream;
  OR_SKIP_AND_RETURN(hipStreamCreate(&stream), "failed to create stream");

  void *src = nullptr;
  void *dst = nullptr;

  OR_SKIP_AND_RETURN(hipMalloc(&src, 100), "");
  OR_SKIP_AND_RETURN(hipMalloc(&dst, 100), "");
  defer(hipFree(src));
  defer(hipFree(dst));

  // create the graph to launch
  OR_SKIP_AND_RETURN(
      hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal), "");
  // copy_launcher(dst, src, 100, stream, launches);
  kernel_launcher(stream, launches);
  OR_SKIP_AND_RETURN(hipStreamEndCapture(stream, &graph), "");
  OR_SKIP_AND_RETURN(hipGraphInstantiate(&instance, graph, NULL, NULL, 0), "");
  defer(hipGraphDestroy(graph));
  defer(hipGraphExecDestroy(instance));

  for (auto _ : state) {
    state.PauseTiming();
    OR_SKIP_AND_BREAK(hipStreamSynchronize(stream), "");
    state.ResumeTiming();
    OR_SKIP_AND_BREAK(hipGraphLaunch(instance, stream), "");
  }

  state.SetItemsProcessed(state.iterations());
  state.counters["cuda_id"] = cuda_id;
  state.counters["numa_id"] = numa_id;
};

static void registerer() {
  for (auto cuda_id : unique_cuda_device_ids()) {
    for (auto numa_id : numa::ids()) {
      std::string name = std::string(NAME) + "/" + std::to_string(numa_id) +
                         "/" + std::to_string(cuda_id);
      benchmark::RegisterBenchmark(
          name.c_str(), Comm_cudart_cudaGraphLaunch, numa_id, cuda_id)
          ->GRAPH_ARGS();
    }
  }
}

SCOPE_AFTER_INIT(registerer, NAME);
