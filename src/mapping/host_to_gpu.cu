#include "hip/hip_runtime.h"
#include <cassert>

#include <hip/hip_runtime.h>
#if USE_NUMA
#include <numa.h>
#endif // USE_NUMA

#include "scope/init/init.hpp"
#include "scope/utils/utils.hpp"
#include "scope/init/flags.hpp"

#include "mapping/args.hpp"
#include "init/flags.hpp"
#include "utils/numa.hpp"

#define NAME "Comm/Mapping/HostToGPU"

#define OR_SKIP(stmt) \
  if (PRINT_IF_ERROR(stmt)) { \
    state.SkipWithError(NAME); \
    return; \
}

template <bool NOOP = false>
__global__ void gpu_write(void *ptr, const size_t count, const size_t stride) {
  if (NOOP) {
    return;
  }

  char *p = (char *) ptr;

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = gx & 31;
  // warp ID
  size_t wx             = gx / 32;
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;

  if (0 == lx) {
    for (size_t i = wx * stride; i < count; i += numWarps * stride) {
      p[i] = 0;
    }
  }
}

static void Comm_Mapping_HostToGPU(benchmark::State &state) {

  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  const size_t pageSize = page_size();

  const auto bytes   = 1ULL << static_cast<size_t>(state.range(0));
  const int dst_cuda  = FLAG(cuda_device_ids)[0];
#if USE_NUMA
  const int src_numa = FLAG(numa_ids)[0];
#endif

#if USE_NUMA
  numa_bind_node(src_numa);
#endif

  OR_SKIP(utils::cuda_reset_device(dst_cuda));
  OR_SKIP(hipSetDevice(dst_cuda));

  void *ptr = aligned_alloc(pageSize, bytes);
  defer(free(ptr));
  std::memset(ptr, 0, bytes);

  OR_SKIP(hipHostRegister(ptr, bytes, hipHostRegisterMapped));
  defer(hipHostUnregister(ptr));

  // get a valid device pointer
  void *dptr;
  hipDeviceProp_t prop;
  OR_SKIP(hipGetDeviceProperties(&prop, dst_cuda));
  if (prop.canUseHostPointerForRegisteredMem) {
    dptr = ptr;
  } else {
    OR_SKIP(hipHostGetDevicePointer(&dptr, ptr, 0));
  }


  hipEvent_t start, stop;
  OR_SKIP(hipEventCreate(&start));
  defer(hipEventDestroy(start));
  OR_SKIP(hipEventCreate(&stop));
  defer(hipEventDestroy(stop));

  for (auto _ : state) {
    hipEventRecord(start);
    gpu_write<<<256, 256>>>(dptr, bytes, 32);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float millis = 0;
    OR_SKIP(hipEventElapsedTime(&millis, start, stop));
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters["bytes"] = bytes;
  state.counters["src_numa"] = src_numa;
  state.counters["dst_cuda"] = dst_cuda;

#if USE_NUMA
  numa_bind_node(-1);
#endif
}

BENCHMARK(Comm_Mapping_HostToGPU)->SMALL_ARGS()->UseManualTime();
