#include "hip/hip_runtime.h"
#include <cassert>

#include <hip/hip_runtime.h>
#if USE_NUMA
#include <numa.h>
#endif // USE_NUMA

#include "scope/init/init.hpp"
#include "scope/utils/utils.hpp"
#include "scope/init/flags.hpp"

#include "mapping/args.hpp"
#include "init/flags.hpp"
#include "utils/numa.hpp"

#define NAME "Comm/Mapping/HostToGPU"

#define OR_SKIP(stmt) \
  if (PRINT_IF_ERROR(stmt)) { \
    state.SkipWithError(NAME); \
    return; \
}

template <typename write_t>
__global__ void gpu_write(write_t *ptr, const size_t bytes) {
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t num_elems = bytes / sizeof(write_t);

  for (size_t i = gx; i < num_elems; i += gridDim.x * blockDim.x) {
    ptr[gx] = 0;
  }
}


template <typename read_t>
__global__ void gpu_read(const read_t *ptr, const size_t bytes) {
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t num_elems = bytes / sizeof(read_t);

  __shared__ int32_t s[256];

  for (size_t i = gx; i < num_elems; i += gridDim.x * blockDim.x) {
    s[threadIdx.x] = ptr[gx];
    (void) s[threadIdx.x];
  }
}


static void Comm_Mapping_HostToGPU(benchmark::State &state, const bool read) {

  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  const size_t pageSize = page_size();

  const auto bytes   = 1ULL << static_cast<size_t>(state.range(0));
  const int dst_cuda  = FLAG(cuda_device_ids)[0];
#if USE_NUMA
  const int src_numa = FLAG(numa_ids)[0];
#endif

#if USE_NUMA
  numa_bind_node(src_numa);
#endif

  OR_SKIP(utils::cuda_reset_device(dst_cuda));
  OR_SKIP(hipSetDevice(dst_cuda));

  void *ptr = aligned_alloc(pageSize, bytes);
  defer(free(ptr));
  std::memset(ptr, 0, bytes);

  OR_SKIP(hipHostRegister(ptr, bytes, hipHostRegisterMapped));
  defer(hipHostUnregister(ptr));

  // get a valid device pointer
  void *dptr;
  hipDeviceProp_t prop;
  OR_SKIP(hipGetDeviceProperties(&prop, dst_cuda));
  if (prop.canUseHostPointerForRegisteredMem) {
    dptr = ptr;
  } else {
    OR_SKIP(hipHostGetDevicePointer(&dptr, ptr, 0));
  }


  hipEvent_t start, stop;
  OR_SKIP(hipEventCreate(&start));
  defer(hipEventDestroy(start));
  OR_SKIP(hipEventCreate(&stop));
  defer(hipEventDestroy(stop));

  for (auto _ : state) {
    hipEventRecord(start);
    if (read) {
      gpu_read<int32_t><<<256, 256>>>((int32_t*) dptr, bytes);
    } else {
      gpu_write<int32_t><<<256, 256>>>((int32_t *)dptr, bytes);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float millis = 0;
    OR_SKIP(hipEventElapsedTime(&millis, start, stop));
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters["bytes"] = bytes;
  state.counters["src_numa"] = src_numa;
  state.counters["dst_cuda"] = dst_cuda;

#if USE_NUMA
  numa_bind_node(-1);
#endif
}

BENCHMARK_CAPTURE(Comm_Mapping_HostToGPU, read, 1)->ARGS()->UseManualTime();
BENCHMARK_CAPTURE(Comm_Mapping_HostToGPU, write, 0)->ARGS()->UseManualTime();