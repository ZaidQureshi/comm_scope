#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>

#include "scope/init/flags.hpp"
#include "scope/init/init.hpp"
#include "scope/utils/utils.hpp"

#include "args.hpp"

#define NAME "Comm_Duplex_MemcpyPeer"

#define OR_SKIP(stmt, msg)                                                                                             \
  if (PRINT_IF_ERROR(stmt)) {                                                                                          \
    state.SkipWithError(msg);                                                                                          \
    return;                                                                                                            \
  }

__global__ void busy_wait2(clock_t *d, clock_t clock_count) {
  clock_t start_clock = clock64();
  clock_t clock_offset = 0;
  while (clock_offset < clock_count) {
    clock_offset = clock64() - start_clock;
  }
  if (d) {
    *d = clock_offset;
  }
}

auto Comm_Duplex_MemcpyPeer = [](benchmark::State &state, const int gpu0, const int gpu1) {
  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  const auto bytes = 1ULL << static_cast<size_t>(state.range(0));

  OR_SKIP(utils::cuda_reset_device(gpu0), NAME " failed to reset src CUDA device");
  OR_SKIP(utils::cuda_reset_device(gpu1), NAME " failed to reset dst CUDA device");

  void *src0 = nullptr;
  void *src1 = nullptr;
  void *dst0 = nullptr;
  void *dst1 = nullptr;
  hipStream_t stream0;
  hipStream_t stream1;
  hipError_t err;
  hipEvent_t start, stop1, stop;

  OR_SKIP(hipSetDevice(gpu0), NAME " failed to set src device");
  OR_SKIP(hipMalloc(&src0, bytes), NAME " failed to perform src0 hipMalloc");
  OR_SKIP(hipMalloc(&dst0, bytes), NAME " failed to perform src1 hipMalloc");
  defer(hipFree(src0));
  defer(hipFree(dst0));
  OR_SKIP(hipMemset(src0, 0, bytes), NAME " failed to perform src0 hipMemset");
  OR_SKIP(hipMemset(dst0, 0, bytes), NAME " failed to perform src1 hipMemset");
  OR_SKIP(hipStreamCreate(&stream0), NAME " failed to create stream");
  defer(hipStreamDestroy(stream0));
  OR_SKIP(hipEventCreate(&start), NAME " couldn't create start event");
  OR_SKIP(hipEventCreate(&stop), NAME " couldn't create stop event");
  defer(hipEventDestroy(start));
  defer(hipEventDestroy(stop));
  if (gpu0 != gpu1) {
    err = hipDeviceDisablePeerAccess(gpu1);
    hipGetLastError(); // clear error
    if (hipSuccess != err && hipErrorPeerAccessNotEnabled != err) {
      state.SkipWithError(NAME " failed to disable peer access");
      return;
    }
  }

  OR_SKIP(hipSetDevice(gpu1), NAME " failed to set dst device");
  OR_SKIP(hipMalloc(&src1, bytes), NAME " failed to perform src1 hipMalloc");
  OR_SKIP(hipMalloc(&dst1, bytes), NAME " failed to perform dst1 hipMalloc");
  defer(hipFree(src1));
  defer(hipFree(dst1));
  OR_SKIP(hipMemset(src1, 0, bytes), NAME " failed to perform dst hipMemset");
  OR_SKIP(hipMemset(dst1, 0, bytes), NAME " failed to perform dst hipMemset");
  OR_SKIP(hipStreamCreate(&stream1), NAME " failed to create stream");
  defer(hipStreamDestroy(stream1));
  OR_SKIP(hipEventCreate(&stop1), NAME " couldn't create stop1 event");
  defer(hipEventDestroy(stop1));
  if (gpu0 != gpu1) {
    err = hipDeviceDisablePeerAccess(gpu0);
    hipGetLastError(); // clear error
    if (hipSuccess != err && hipErrorPeerAccessNotEnabled != err) {
      state.SkipWithError(NAME " failed to disable peer access");
      return;
    }
  }
  

  size_t cycles = 4096;
  for (auto _ : state) {
    OR_SKIP(hipSetDevice(gpu0), NAME " failed to set src device");
    busy_wait2<<<1,1, 0, stream0>>>(nullptr, cycles);
    OR_SKIP(hipGetLastError(), NAME " failed to busy_wait");
    OR_SKIP(hipEventRecord(start, stream0), NAME " failed to record start");
    OR_SKIP(hipMemcpyPeerAsync(dst1, gpu1, src0, gpu0, bytes, stream0), NAME " failed to memcpy");
    OR_SKIP(hipSetDevice(gpu1), NAME " failed to set src device");
    OR_SKIP(hipMemcpyPeerAsync(dst0, gpu0, src1, gpu1, bytes, stream1), NAME " failed to memcpy");
    OR_SKIP(hipEventRecord(stop1, stream1), NAME " failed to stop");
    OR_SKIP(hipSetDevice(gpu0), NAME " failed to set src device");
    OR_SKIP(hipStreamWaitEvent(stream0, stop1, 0), NAME " failed to set src device");
    OR_SKIP(hipEventRecord(stop, stream0), NAME " failed to stop");

    // if kernel has ended, it wasn't long enough to cover the host code.
    // finish transfers, increase cycles, and try again
    err = hipEventQuery(start);
    if (hipSuccess == err) {
      cycles *= 1.5;
      OR_SKIP(hipStreamSynchronize(stream0), NAME " failed to wait for stream0");
      OR_SKIP(hipStreamSynchronize(stream1), NAME " failed to wait for stream1");
      continue;
    } else if (hipErrorNotReady == err) {
      // kernel was long enough
    } else {
     OR_SKIP(err, NAME " errored while waiting for kernel");
    }

    OR_SKIP(hipEventSynchronize(stop), NAME " failed to synchronize");

    float ms = 0.0f;
    OR_SKIP(hipEventElapsedTime(&ms, start, stop), NAME "failed to compute elapsed time");
    state.SetIterationTime(ms / 1000);
  }
  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes) * 2);
  state.counters["bytes"]  = bytes;
  state.counters["gpu0"] = gpu0;
  state.counters["gpu1"] = gpu1;
  state.counters["wait_cycles"] = cycles;
};

static void registerer() {
  std::string name;
  for (size_t i = 0; i < unique_cuda_device_ids().size(); ++i) {
    for (size_t j = i; j < unique_cuda_device_ids().size(); ++j) {
      auto gpu0 = unique_cuda_device_ids()[i];
      auto gpu1 = unique_cuda_device_ids()[j];
      name        = std::string(NAME) + "/" + std::to_string(gpu0) + "/" + std::to_string(gpu1);
      benchmark::RegisterBenchmark(name.c_str(), Comm_Duplex_MemcpyPeer, gpu0, gpu1)->SMALL_ARGS()->UseManualTime();
    }
  }
}

SCOPE_REGISTER_AFTER_INIT(registerer, NAME);
