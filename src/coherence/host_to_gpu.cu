
#include <hip/hip_runtime.h>
#if CUDA_VERSION_MAJOR >= 8

#include <assert.h>
#include <iostream>
#include <stdio.h>
#include <string.h>

#include <cuda_runtime.h>
#if USE_NUMA
#include <numa.h>
#endif

#include "scope/init/init.hpp"
#include "scope/utils/utils.hpp"
#include "scope/init/flags.hpp"

#include "args.hpp"
#include "init/flags.hpp"

#define NAME "Comm/UM/Coherence/HostToGPU"

template <bool NOOP = false>
__global__ void gpu_write(char *ptr, const size_t count, const size_t stride) {
  if (NOOP) {
    return;
  }

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = gx & 31;
  // warp ID
  size_t wx             = gx / 32;
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;

  if (0 == lx) {
    for (size_t i = wx * stride; i < count; i += numWarps * stride) {
      ptr[i] = 0;
    }
  }
}

static void Comm_UM_Coherence_HostToGPU(benchmark::State &state) {

  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  const size_t pageSize = page_size();

  const auto bytes   = 1ULL << static_cast<size_t>(state.range(0));
  const int dst_gpu  = FLAG(cuda_device_ids)[0];
#if USE_NUMA
  const int src_numa = FLAG(numa_ids)[0];
#endif

#if USE_NUMA
  numa_bind_node(src_numa);
#endif

  if (PRINT_IF_ERROR(utils::cuda_reset_device(dst_gpu))) {
    state.SkipWithError(NAME " failed to reset device");
    return;
  }

  if (PRINT_IF_ERROR(cudaSetDevice(dst_gpu))) {
    state.SkipWithError(NAME " failed to set CUDA dst device");
    return;
  }

  char *ptr = nullptr;
  if (PRINT_IF_ERROR(cudaMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMallocManaged");
    return;
  }
  defer(cudaFree(ptr));

  if (PRINT_IF_ERROR(cudaMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMemset");
    return;
  }

  cudaEvent_t start, stop;
  if (PRINT_IF_ERROR(cudaEventCreate(&start))) {
    state.SkipWithError(NAME " failed to create start event");
    return;
  }
  defer(cudaEventDestroy(start));

  if (PRINT_IF_ERROR(cudaEventCreate(&stop))) {
    state.SkipWithError(NAME " failed to create end event");
    return;
  }
  defer(cudaEventDestroy(stop));

  for (auto _ : state) {
    cudaError_t err = cudaMemPrefetchAsync(ptr, bytes, cudaCpuDeviceId);
    if (err == cudaErrorInvalidDevice) {
      for (size_t i = 0; i < bytes; i += pageSize) {
        ptr[i] = 0;
      }
    }

    if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }

    cudaEventRecord(start);
    gpu_write<<<256, 256>>>(ptr, bytes, pageSize);
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);

    float millis = 0;
    if (PRINT_IF_ERROR(cudaEventElapsedTime(&millis, start, stop))) {
      state.SkipWithError(NAME " failed to get elapsed time");
      break;
    }
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters.insert({{"bytes", bytes}});

#if USE_NUMA
  numa_bind_node(-1);
#endif
}

BENCHMARK(Comm_UM_Coherence_HostToGPU)->SMALL_ARGS()->UseManualTime();

#endif // CUDA_VERSION_MAJOR >= 8