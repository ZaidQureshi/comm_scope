
#include <hip/hip_runtime.h>
#if CUDA_VERSION_MAJOR >= 8

#include <cassert>

#include <cuda_runtime.h>
#if USE_NUMA
#include <numa.h>
#endif // USE_NUMA

#include "scope/init/init.hpp"
#include "scope/utils/utils.hpp"
#include "scope/init/flags.hpp"

#include "args.hpp"
#include "init/flags.hpp"
#include "utils/numa.hpp"
#include "init/numa.hpp"
#include "utils/cache_control.hpp"

#define NAME "Comm_Coherence_Duplex_HostGPU"

#define OR_SKIP(stmt) \
  if (PRINT_IF_ERROR(stmt)) { \
    state.SkipWithError(NAME); \
    return; \
}

template <bool NOOP = false>
__global__ void gpu_write(char *ptr, const size_t count, const size_t stride) {
  if (NOOP) {
    return;
  }

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = gx & 31;
  // warp ID
  size_t wx             = gx / 32;
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;

  if (0 == lx) {
    for (size_t i = wx * stride; i < count; i += numWarps * stride) {
      ptr[i] = 0;
    }
  }
}

auto Comm_Coherence_Duplex_HostGPU = [] (benchmark::State &state,
  #if USE_NUMA
  const int numa_id,
  #endif // USE_NUMA
  const int cuda_id) {

  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  const size_t pageSize = page_size();
  const auto bytes   = 1ULL << static_cast<size_t>(state.range(0));

#if USE_NUMA
  numa_bind_node(numa_id);
#endif

  OR_SKIP(utils::cuda_reset_device(cuda_id));
  OR_SKIP(cudaSetDevice(cuda_id));

  char *ptrs[2] = {nullptr};
  OR_SKIP(cudaMallocManaged(&ptrs[0], bytes));
  OR_SKIP(cudaMallocManaged(&ptrs[1], bytes));
  OR_SKIP(cudaMemset(ptrs[0], 0, bytes));
  OR_SKIP(cudaMemset(ptrs[1], 0, bytes));
  OR_SKIP(cudaDeviceSynchronize());
  

  for (auto _ : state) {
    state.PauseTiming();

    cudaError_t err;
    // move ptrs[0] to cpu
    err = cudaMemPrefetchAsync(ptrs[0], bytes, cudaCpuDeviceId);
    if (err == cudaErrorInvalidDevice) {
      for (size_t i = 0; i < bytes; i += pageSize) {
        ptrs[0][i] = 0;
      }
    }
    flush_all(ptrs[0], bytes);


    // move ptrs[1] to gpu
    err = cudaMemPrefetchAsync(ptrs[1], bytes, cuda_id);
    if (err == cudaErrorInvalidDevice) {
      gpu_write<<<256, 256>>>(ptrs[1], bytes, pageSize);
    }
    OR_SKIP(cudaDeviceSynchronize());

    state.ResumeTiming();
    // access ptrs[0] on gpu
    gpu_write<<<256, 256>>>(ptrs[0], bytes, pageSize);
    // access prts[1] on cpu
    for (size_t i = 0; i < bytes; i += pageSize) {
      ptrs[1][i] = 0;
    }
    OR_SKIP(cudaDeviceSynchronize());
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes) * 2);
  state.counters["bytes"] = bytes;
  state.counters["cuda_id"] = cuda_id;
#if USE_NUMA
  state.counters["numa_id"] = numa_id;
#endif // USE_NUMA

#if USE_NUMA
  numa_bind_node(-1);
#endif

  for (auto p : ptrs) {
    OR_SKIP(cudaFree(p));
  }

};

static void registerer() {
  for (auto cuda_id : unique_cuda_device_ids()) {

    cudaDeviceProp prop;
    cudaGetDeviceProperties(&prop, cuda_id);
    if (!prop.concurrentManagedAccess) {
      LOG(debug, "{} can't run on device {}: requires concurrent managed access", NAME, cuda_id);
      continue;
    }

#if USE_NUMA
    for (auto numa_id : unique_numa_ids()) {
#endif // USE_NUMA
      std::string name = std::string(NAME)
#if USE_NUMA 
                       + "/" + std::to_string(numa_id) 
#endif // USE_NUMA
                       + "/" + std::to_string(cuda_id);
      benchmark::RegisterBenchmark(name.c_str(), Comm_Coherence_Duplex_HostGPU,
#if USE_NUMA
        numa_id,
#endif // USE_NUMA
        cuda_id)->SMALL_ARGS();
#if USE_NUMA
    }
#endif // USE_NUMA
  }
}

SCOPE_REGISTER_AFTER_INIT(registerer);

#endif // CUDA_VERSION_MAJOR >= 8
