
#include <hip/hip_runtime.h>
#if CUDA_VERSION_MAJOR >= 8

#include <cassert>

#include <cuda_runtime.h>
#if USE_NUMA
#include <numa.h>
#endif // USE_NUMA

#include "scope/init/init.hpp"
#include "scope/utils/utils.hpp"
#include "scope/init/flags.hpp"

#include "args.hpp"
#include "init/flags.hpp"
#include "utils/numa.hpp"

#define NAME "Comm/UM/Latency/GPUToHost"

template <bool NOOP = false>
void cpu_traverse(size_t *ptr, const size_t steps) {

  if (NOOP) {
    return;
  }
  size_t next = 0;
  for (size_t i = 0; i < steps; ++i) {
    next = ptr[next];
  }
  ptr[next] = 1;
}

static void Comm_UM_Latency_GPUToHost(benchmark::State &state) {

  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  const size_t steps = state.range(0);
  const int cuda_id  = FLAG(cuda_device_ids)[0];
#if USE_NUMA
  const int numa_id  = FLAG(numa_ids)[0];
#endif

  const size_t stride = 65536 * 2;
  const size_t bytes  = sizeof(size_t) * (steps + 1) * stride;
#if USE_NUMA
  numa_bind_node(numa_id);
#endif
  if (PRINT_IF_ERROR(cudaSetDevice(cuda_id))) {
    state.SkipWithError(NAME " failed to set CUDA device");
    return;
  }
  if (PRINT_IF_ERROR(cudaDeviceReset())) {
    state.SkipWithError(NAME " failed to reset device");
    return;
  }

  size_t *ptr = nullptr;
  if (PRINT_IF_ERROR(cudaMallocManaged(&ptr, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMallocManaged");
    return;
  }
  defer(cudaFree(ptr));

  if (PRINT_IF_ERROR(cudaMemset(ptr, 0, bytes))) {
    state.SkipWithError(NAME " failed to perform cudaMemset");
    return;
  }

  // set up stride pattern
  for (size_t i = 0; i < steps; ++i) {
    ptr[i * stride] = (i + 1) * stride;
  }
  if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
    state.SkipWithError(NAME " failed to synchronize");
    return;
  }

  for (auto _ : state) {
    state.PauseTiming();
    // prefetch to source
    if (PRINT_IF_ERROR(cudaMemPrefetchAsync(ptr, bytes, cuda_id))) {
      state.SkipWithError(NAME " failed prefetch to device");
      return;
    }
    if (PRINT_IF_ERROR(cudaDeviceSynchronize())) {
      state.SkipWithError(NAME " failed to synchronize");
      return;
    }
    state.ResumeTiming();

    cpu_traverse(ptr, steps);
  }
  state.counters["strides"] = steps;

#if USE_NUMA
  // reset to run on any node
  numa_bind_node(-1);
#endif
}

BENCHMARK(Comm_UM_Latency_GPUToHost)->SMALL_ARGS()->MinTime(0.1);

#endif // CUDA_VERSION_MAJOR >= 8