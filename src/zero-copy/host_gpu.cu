#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#if USE_NUMA
#include <numa.h>
#endif // USE_NUMA

#include "scope/init/init.hpp"
#include "scope/utils/utils.hpp"
#include "scope/init/flags.hpp"

#include "zero-copy/args.hpp"
#include "init/flags.hpp"
#include "utils/numa.hpp"
#include "init/numa.hpp"
#include "utils/cache_control.hpp"


#define NAME "Comm_ZeroCopy_HostGPU"

#define OR_SKIP(stmt) \
  if (PRINT_IF_ERROR(stmt)) { \
    state.SkipWithError(NAME); \
    return; \
}

typedef enum {
  READ,
  WRITE,
} AccessType;

std::string to_string(const AccessType &a) {
  if (a == READ) {
    return "_Read";
  } else {
    return "_Write";
  }
}

// typedef enum {
//   FLUSH,
//   NO_FLUSH,
// } FlushType;

// static std::string to_string(const FlushType &a) {
//   if (a == FLUSH) {
//     return "_Flush";
//   } else {
//     return "";
//   }
// }

template <typename write_t>
__global__ void gpu_write(write_t *ptr, const size_t bytes) {
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t num_elems = bytes / sizeof(write_t);

  for (size_t i = gx; i < num_elems; i += gridDim.x * blockDim.x) {
    ptr[i] = 0;
  }
}


template <typename read_t>
__global__ void gpu_read(const read_t *ptr, const size_t bytes) {
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t num_elems = bytes / sizeof(read_t);

  __shared__ int32_t s[256];
  int32_t t;

  for (size_t i = gx; i < num_elems; i += gridDim.x * blockDim.x) {
    t += ptr[i];
  }
  s[threadIdx.x] = t;
  (void) s[threadIdx.x];
}


auto Comm_ZeroCopy_HostGPU = [](benchmark::State &state, const int src_numa, const int dst_cuda, const AccessType access_type) {

  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  const size_t pageSize = page_size();

  const auto bytes   = 1ULL << static_cast<size_t>(state.range(0));

#if USE_NUMA
  numa_bind_node(src_numa);
#endif

  OR_SKIP(utils::cuda_reset_device(dst_cuda));
  OR_SKIP(hipSetDevice(dst_cuda));

  void *ptr = aligned_alloc(pageSize, bytes);
  defer(free(ptr));
  if (!ptr && bytes) {
    state.SkipWithError(NAME " failed to allocate host memory");
    return;
  }
  std::memset(ptr, 0, bytes);

  OR_SKIP(hipHostRegister(ptr, bytes, hipHostRegisterMapped));
  defer(hipHostUnregister(ptr));

  // get a valid device pointer
  void *dptr;
  hipDeviceProp_t prop;
  OR_SKIP(hipGetDeviceProperties(&prop, dst_cuda));
  if (prop.canUseHostPointerForRegisteredMem) {
    dptr = ptr;
  } else {
    OR_SKIP(hipHostGetDevicePointer(&dptr, ptr, 0));
  }

  hipEvent_t start, stop;
  OR_SKIP(hipEventCreate(&start));
  defer(hipEventDestroy(start));
  OR_SKIP(hipEventCreate(&stop));
  defer(hipEventDestroy(stop));

  for (auto _ : state) {

    OR_SKIP(hipEventRecord(start));
    if (READ == access_type) {
      gpu_read<int32_t><<<256, 256>>>((int32_t*) dptr, bytes);
    } else {
      gpu_write<int32_t><<<256, 256>>>((int32_t *)dptr, bytes);
    }

    OR_SKIP(hipEventRecord(stop));
    OR_SKIP(hipEventSynchronize(stop));

    float millis = 0;
    OR_SKIP(hipEventElapsedTime(&millis, start, stop));
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes));
  state.counters["bytes"] = bytes;
#if USE_NUMA
  state.counters["src_numa"] = src_numa;
#endif // USE_NUMA
  state.counters["dst_cuda"] = dst_cuda;

#if USE_NUMA
  numa_bind_node(-1);
#endif
};

static void registerer() {

  for (auto workload : {READ, WRITE}) {



  for (auto cuda_id : unique_cuda_device_ids()) {
#if USE_NUMA
    for (auto numa_id : unique_numa_ids()) {
#endif // USE_NUMA
      std::string name = std::string(NAME) + to_string(workload)
#if USE_NUMA 
                       + "/" + std::to_string(numa_id) 
#endif // USE_NUMA
                       + "/" + std::to_string(cuda_id);
      benchmark::RegisterBenchmark(name.c_str(), Comm_ZeroCopy_HostGPU,
#if USE_NUMA
        numa_id,
#endif // USE_NUMA
        cuda_id, workload)->ARGS()->UseManualTime();
#if USE_NUMA
    }
#endif // USE_NUMA
  }
}
}

SCOPE_REGISTER_AFTER_INIT(registerer);
