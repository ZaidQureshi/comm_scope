
#include <hip/hip_runtime.h>
#if __CUDACC_VER_MAJOR__ >= 8

#include <cassert>

#include <cuda_runtime.h>

#include "scope/init/flags.hpp"
#include "scope/init/init.hpp"
#include "scope/utils/utils.hpp"

#include "args.hpp"

#define NAME "Comm_Demand_Duplex_GPUGPU"

#define OR_SKIP(stmt)                                                                                                  \
  if (PRINT_IF_ERROR(stmt)) {                                                                                          \
    state.SkipWithError(NAME);                                                                                         \
    return;                                                                                                            \
  }

template <bool NOOP = false>
__global__ void gpu_write(char *ptr, const size_t count, const size_t stride) {
  if (NOOP) {
    return;
  }

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = gx & 31;
  // warp ID
  size_t wx             = gx / 32;
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;

  if (0 == lx) {
    for (size_t i = wx * stride; i < count; i += numWarps * stride) {
      ptr[i] = 0;
    }
  }
}

auto Comm_Demand_Duplex_GPUGPU = [](benchmark::State &state, const int gpu0, const int gpu1) {
  if (!has_cuda) {
    state.SkipWithError(NAME " no CUDA device found");
    return;
  }

  if (gpu0 == gpu1) {
    state.SkipWithError(NAME " requuires two different GPUs");
    return;
  }

  const size_t pageSize   = page_size();
  const auto bytes        = 1ULL << static_cast<size_t>(state.range(0));
  cudaStream_t streams[2] = {nullptr};
  char *ptrs[2]           = {nullptr};

  // start and end events in gpu0's stream. end0 will not be recorded until after end1
  cudaEvent_t start = nullptr;
  cudaEvent_t end0  = nullptr;

  // end event
  cudaEvent_t end1 = nullptr;

  // initialize data structures for device `dev`
#define INIT(dev)                                                                                                      \
  OR_SKIP(utils::cuda_reset_device(gpu##dev));                                                                         \
  OR_SKIP(cudaSetDevice(gpu##dev));                                                                                    \
  OR_SKIP(cudaStreamCreate(&streams[dev]));                                                                            \
  OR_SKIP(cudaMallocManaged(&ptrs[dev], bytes));                                                                       \
  OR_SKIP(cudaMemset(ptrs[dev], 0, bytes))

  INIT(0);
  INIT(1);

  // record the "pimary" events in the stream associated with gpu0
  OR_SKIP(cudaSetDevice(gpu0));
  OR_SKIP(cudaEventCreate(&start))
  OR_SKIP(cudaEventCreate(&end0))

  // record the end of the transfer task running on gpu1
  OR_SKIP(cudaSetDevice(gpu1));
  OR_SKIP(cudaEventCreate(&end1))

  for (auto _ : state) {
    // prefetch data to the source device before the transfers
    OR_SKIP(cudaMemPrefetchAsync(ptrs[0], bytes, gpu1, streams[0]));
    OR_SKIP(cudaMemPrefetchAsync(ptrs[1], bytes, gpu0, streams[1]));
    OR_SKIP(cudaStreamSynchronize(streams[0]));
    OR_SKIP(cudaStreamSynchronize(streams[1]));

    OR_SKIP(cudaSetDevice(gpu0));
    OR_SKIP(cudaEventRecord(start, streams[0]));
    gpu_write<<<256, 256, 0, streams[0]>>>(ptrs[0], bytes, pageSize);
    OR_SKIP(cudaGetLastError());
    OR_SKIP(cudaSetDevice(gpu1));
    gpu_write<<<256, 256, 0, streams[1]>>>(ptrs[1], bytes, pageSize);
    OR_SKIP(cudaGetLastError());
    OR_SKIP(cudaEventRecord(end1, streams[1]));
    OR_SKIP(cudaSetDevice(gpu0));
    OR_SKIP(cudaStreamWaitEvent(streams[0], end1, 0 /*must be 0*/));
    OR_SKIP(cudaEventRecord(end0, streams[0]));

    // once stream 0 is finished, we can compute the elapsed time
    OR_SKIP(cudaStreamSynchronize(streams[0]));
    float millis = 0;
    OR_SKIP(cudaEventElapsedTime(&millis, start, end0));
    state.SetIterationTime(millis / 1000);
  }

  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(bytes) * 2);
  state.counters["bytes"] = bytes;
  state.counters["gpu0"]  = gpu0;
  state.counters["gpu1"]  = gpu1;

  OR_SKIP(cudaEventDestroy(start));
  OR_SKIP(cudaEventDestroy(end0));
  OR_SKIP(cudaEventDestroy(end1));

  for (auto s : streams) {
    OR_SKIP(cudaStreamDestroy(s));
  }

  for (auto p : ptrs) {
    OR_SKIP(cudaFree(p));
  }
};

static void registerer() {
  for (size_t i : unique_cuda_device_ids()) {
    for (size_t j : unique_cuda_device_ids()) {
      if (i < j) {
        std::string name = std::string(NAME) + "/" + std::to_string(i) + "/" + std::to_string(j);
        benchmark::RegisterBenchmark(name.c_str(), Comm_Demand_Duplex_GPUGPU, i, j)->SMALL_ARGS()->UseManualTime();
      }
    }
  }
}

SCOPE_REGISTER_AFTER_INIT(registerer, NAME);

#endif // __CUDACC_VER_MAJOR__ >= 8
